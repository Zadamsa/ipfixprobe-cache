
#include "cudaPacketBlock.cuh"

namespace ipxp {


PacketBlock* getCudaPacketBlock(size_t pkts_size)
{
    //Packet* packets = nullptr;
    //hipHostAlloc(&packets, sizeof(Packet) * pkts_size, hipHostMallocMapped);
    auto res = new PacketBlock(pkts_size);
    for (size_t i = 0; i < pkts_size; ++i) {
        hipHostAlloc((void**)&res->pkts[i].packet, 256, hipHostMallocMapped);
        hipHostGetDevicePointer((void**)&res->pkts[i].packet_dev, (void*)res->pkts[i].packet, 0);
    }
    return res;
}

void freeCudaPacketBlock(PacketBlock* block)
{
    hipHostFree(block->pkts);
    block->pkts = nullptr;
}

} // namespace ipxp